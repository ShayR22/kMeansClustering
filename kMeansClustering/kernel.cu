#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "cudaHeader.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define CPU_TO_GPU hipMemcpyHostToDevice
#define GPU_TO_CPU hipMemcpyDeviceToHost
#define WRAP_SIZE 32
#define NUM_OF_THREADS_PER_BLOCK 256

#define RETURN_IF_NOT_TRUE(truth)if(!truth) {return 0;}

static int setDevice();

static int allocate_GPU_space(void** data, int dataCount, int dataTypeSize);
static int free_GPU_space(void* data);

static int copy_to_GPU_memory(void* copyTo, void* copyFrom, int count, int typeSize);
static int copy_from_GPU_memory(void* copyTo, void* copyFrom, int count, int typeSize);

static int checkError(hipError_t error);

__device__ double getDistance(vector_t *v1, vector_t *v2)
{
	int i;
	double sumOfDeltaSquares = 0;
	double delta;
	for (i = 0; i < NUM_OF_ELEMENTS_IN_VECTOR; i++)
	{
		delta = v1->axis[i] - v2->axis[i];
		sumOfDeltaSquares += delta * delta;
	}
	return sqrt(sumOfDeltaSquares);
}


//increment points START
__device__ void advance_point_in_deltaT(point_t *p, double deltaT)
{
	int i;
	for (i = 0; i < NUM_OF_ELEMENTS_IN_VECTOR; i++)
	{
		p->location.axis[i] += p->speed.axis[i] * deltaT;
	}
}

__global__ void increment_points(point_t *devPoints, int pointsCount, double deltaT)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < pointsCount)
	{
		point_t *point = &(devPoints[index]);
		advance_point_in_deltaT(point, deltaT);
	}
}


int cuda_increment_points(point_t *points, int pointsCount, double deltaT)
{
	point_t *devPoints;
	setDevice();
	// allocate space and then copy points in to it
	RETURN_IF_NOT_TRUE(allocate_GPU_space((void**)&devPoints, pointsCount, sizeof(point_t)));
	RETURN_IF_NOT_TRUE(copy_to_GPU_memory(devPoints, points, pointsCount, sizeof(point_t)));

	int numOfBlocks = pointsCount / NUM_OF_THREADS_PER_BLOCK + 1;
	increment_points <<<numOfBlocks, NUM_OF_THREADS_PER_BLOCK >>> (devPoints, pointsCount, deltaT);
	RETURN_IF_NOT_TRUE(copy_from_GPU_memory(points, devPoints, pointsCount, sizeof(point_t)));

	RETURN_IF_NOT_TRUE(free_GPU_space(devPoints));
	return 1;
}
//increment points END


// calcPointsNearestCluster START
__global__ void calcPointNearestCluster(point_t *devPoints, int pointsCount, cluster_t *devClusters, int clustersCount)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < pointsCount)
	{
		vector_t *pointLocation = &(devPoints[index].location);
		vector_t *currentClusterLocation;
		cluster_t *nearestCluster = NULL;

		double distance = INT_MAX;
		double currentDistance;

		int i;
		for (i = 0; i < clustersCount; i++)
		{
			currentClusterLocation = &(devClusters[i].center);
			currentDistance = getDistance(pointLocation, currentClusterLocation);
			if(currentDistance < distance)
			{
				distance = currentDistance;
				nearestCluster = &(devClusters[i]);
			}
		}
		devPoints[index].clusterBelongTo = nearestCluster->id;
	}
}

int cuda_add_to_each_point_its_nearest_cluster(point_t *points, int pointsCount, cluster_t *clusters, int clustersCount)
{
	setDevice();
	
	cluster_t *devClusters;
	point_t *devPoints;
		
	// allocate GPU space
	RETURN_IF_NOT_TRUE(allocate_GPU_space((void**)&devClusters, clustersCount, sizeof(cluster_t)));
	RETURN_IF_NOT_TRUE(allocate_GPU_space((void**)&devPoints, pointsCount, sizeof(point_t)));
	
	//copy clusters and point to GPU's memory
	RETURN_IF_NOT_TRUE(copy_to_GPU_memory(devClusters, clusters, clustersCount, sizeof(cluster_t)));
	RETURN_IF_NOT_TRUE(copy_to_GPU_memory(devPoints, points, pointsCount, sizeof(point_t)));
		
	int numBlocks = pointsCount / NUM_OF_THREADS_PER_BLOCK + 1;
	calcPointNearestCluster <<<numBlocks, NUM_OF_THREADS_PER_BLOCK>>> (devPoints, pointsCount, devClusters, clustersCount);
		
	RETURN_IF_NOT_TRUE(copy_from_GPU_memory(points, devPoints, pointsCount, sizeof(point_t)));
		
	RETURN_IF_NOT_TRUE(free_GPU_space(devClusters));
	RETURN_IF_NOT_TRUE(free_GPU_space(devPoints));
	
	return 1;
}
// calcPointsNearestCluster END

// calcDiamaters START

__global__ void calc_diameters(clusterHelper_t *devHelpers, int *devHelpersIDs ,point_t *devPoints)
{
	int helperNumber = blockDim.x * blockIdx.x;
	clusterHelper_t *helper = &(devHelpers[helperNumber]);
	int numPoints = helper->numOfPoints;

	vector_t *v1;
	vector_t *v2;
	point_t *p1;
	point_t *p2;
	int *currentIDS;
	int sumIDs = 0;

	double maxDistance = 0;
	double currentDistance;

	int i, j;
	for (i = 0; i < numPoints - 1; i++)
	{
		currentIDS = &(devHelpersIDs[sumIDs]);
		p1 = &(devPoints[currentIDS[i]]);
		v1 = &(p1->location);
		for (j = i + 1; j < numPoints; j++)
		{
			p2 = &(devPoints[currentIDS[j]]);
			v2 = &(p2->location);
			currentDistance = getDistance(v1, v2);

			if (currentDistance > maxDistance)
			{
				maxDistance = currentDistance;
			}
		}
		sumIDs += devHelpers[i].numOfPoints;
	}
	helper->maxDistancePoint = maxDistance;
}

//TODO this work with small sample data but not with big data
int cuda_calc_each_helper_diamater(point_t *points, int pointsCount, clusterHelper_t *helpers, int helpersCount)
{
	setDevice();
	
	clusterHelper_t *devHelpers;
	point_t *devPoints;
	int *devHelpersIDs;
	
	// allocate GPU space
	RETURN_IF_NOT_TRUE(allocate_GPU_space((void**)&devHelpers, helpersCount, sizeof(clusterHelper_t)));
	RETURN_IF_NOT_TRUE(allocate_GPU_space((void**)&devPoints, pointsCount, sizeof(point_t)));
	
	//copy helpers and point to GPU's memory
	RETURN_IF_NOT_TRUE(copy_to_GPU_memory(devPoints, points, pointsCount, sizeof(point_t)));
	RETURN_IF_NOT_TRUE(copy_to_GPU_memory(devHelpers, helpers, helpersCount, sizeof(clusterHelper_t)));

	RETURN_IF_NOT_TRUE(allocate_GPU_space((void**)&devHelpersIDs, pointsCount, sizeof(int)));


	//copy to devHelpersIDS all points IDS from the helpers;
	int k;
	int numPoints;
	int currentSum = 0;
	clusterHelper_t *currentHelper;
	for (k = 0; k < helpersCount; k++)
	{
		currentHelper = &(helpers[k]);
		numPoints = currentHelper->numOfPoints;
		RETURN_IF_NOT_TRUE(copy_to_GPU_memory(&(devHelpersIDs[currentSum]), currentHelper->pointsIDs, numPoints, sizeof(int)));		
		currentSum += numPoints;
	}

	int numBlocks = helpersCount;
	calc_diameters << <numBlocks, 1 >> > (devHelpers, devHelpersIDs ,devPoints);

	RETURN_IF_NOT_TRUE(copy_from_GPU_memory(helpers, devHelpers, helpersCount, sizeof(clusterHelper_t)));

	RETURN_IF_NOT_TRUE(free_GPU_space(devHelpersIDs));
	RETURN_IF_NOT_TRUE(free_GPU_space(devHelpers));
	RETURN_IF_NOT_TRUE(free_GPU_space(devPoints));

	return 1;
}

// calcDiamaters END

static int setDevice()
{
	int device = 0;
	hipError_t error = hipSetDevice(device);
	return checkError(error);
}

static int allocate_GPU_space(void** data, int dataCount, int dataTypeSize)
{
	hipError_t error = hipMalloc(data, dataCount * dataTypeSize);
	return checkError(error);
}

static int free_GPU_space(void* data)
{
	hipError_t error = hipFree(data);
	return checkError(error);
}

static int copy_to_GPU_memory(void* copyTo, void* copyFrom, int count, int typeSize)
{
	hipError_t error = hipMemcpy(copyTo, copyFrom, count*typeSize, CPU_TO_GPU);
	return checkError(error);
}

static int copy_from_GPU_memory(void *copyTo, void* copyFrom, int count, int typeSize)
{
	hipError_t error = hipMemcpy(copyTo, copyFrom, count*typeSize, GPU_TO_CPU);
	return checkError(error);
}


static int checkError(hipError_t error)
{
	if (error != hipSuccess)
	{
		printf("error number %d occured\n", error);
		fflush(stdout);
		return 0;
	}
	return 1;
}

// calcHelperPoints START
//__global__ void calcHelpersPoints(clusterHelper_t *devHelpers, int *devHelpersPointArray, point_t *devPoints, int pointsCount)
//{
//	int helperNumber = blockDim.x * blockIdx.x;
//	int *numOfPoints = &(devHelpersPointArray[helperNumber * pointsCount]);
//
//	int i;
//	for (i = 0; i < pointsCount; i++)
//	{
//		if (devPoints[i].clusterBelongTo == helperNumber)
//		{
//			numOfPoints[i] = devPoints[i].clusterBelongTo;
//			devHelpers[helperNumber].numOfPoints++;
//		}
//	}
//}
//
//
//int cuda_add_to_each_clusterHelper_its_points(clusterHelper_t *helpers, int helpersCount, point_t *points, int pointsCount)
//{
//	setDevice();
//
//	clusterHelper_t *devHelpers;
//	int *devHelpersPointsArray;
//	point_t *devPoints;
//
//	// allocate GPU space
//	RETURN_IF_NOT_TRUE(allocate_GPU_space((void**)&devHelpers, helpersCount, sizeof(clusterHelper_t)));
//	RETURN_IF_NOT_TRUE(allocate_GPU_space((void**)&devHelpersPointsArray, helpersCount * pointsCount, sizeof(int)));
//	RETURN_IF_NOT_TRUE(allocate_GPU_space((void**)&devPoints, pointsCount, sizeof(point_t)));
//
//	//copy clusters and point to GPU's memory
//	RETURN_IF_NOT_TRUE(copy_to_GPU_memory(devPoints, points, pointsCount, sizeof(point_t)));
//	RETURN_IF_NOT_TRUE(copy_to_GPU_memory(devHelpers, helpers, helpersCount, sizeof(clusterHelper_t)));
//
//
//	int numBlocks = helpersCount;
//	calcHelpersPoints <<<numBlocks, 1>>> (devHelpers, devHelpersPointsArray ,devPoints, pointsCount);
//
//
//
//	RETURN_IF_NOT_TRUE(copy_from_GPU_memory(helpers, devHelpers, helpersCount, sizeof(clusterHelper_t)));
//	int i;
//	for (i = 0; i < helpersCount; i++)
//	{
//		RETURN_IF_NOT_TRUE(copy_from_GPU_memory(helpers[i].pointsIDs, &(devHelpersPointsArray[i*pointsCount]), pointsCount, sizeof(int)));
//	}
//
//	RETURN_IF_NOT_TRUE(free_GPU_space(devHelpers));
//	RETURN_IF_NOT_TRUE(free_GPU_space(devPoints));
//	RETURN_IF_NOT_TRUE(free_GPU_space(devHelpersPointsArray));
//
//	return 1;
//
//}
// calcHelperPoints END


//__global__ void calcPointCluster(cluster_t *devClusters, int clusterCount, point_t *devPoints, int pointCount)
//{
//	int offset = blockIdx.x * blockDim.x + threadIdx.x;
//	if (offset < pointCount) // pointCount doesnt have to be divisble by 256
//	{
//		point_t *point = &(devPoints[offset]);
//		int clusterNumber = -1;
//		double minDistance = INT_MAX;
//		double currentDistance;
//		vector_t *currentClusterCenter;
//
//		//iterate on all cluster for the point and save the cluster number and its distance from the point.
//		int i, j;
//		double sumOfDeltas = 0;
//		double currentDelta;
//		for (i = 0; i < clusterCount; i++)
//		{
//			currentClusterCenter = &(devClusters[i].center);
//
//			//TODO put in method
//			//calculate the distance between the point location and the cluster center
//			sumOfDeltas = 0;
//			for (j = 0; j < NUM_OF_ELEMENTS_IN_VECTOR; j++)
//			{
//				currentDelta = currentClusterCenter->axis[j] - point->location.axis[j];
//				sumOfDeltas += currentDelta * currentDelta;
//			}
//			currentDistance = sqrt(sumOfDeltas);
//
//			//update relevant data for the point if 
//			if (currentDistance < minDistance)
//			{
//				minDistance = currentDistance;
//				clusterNumber = i;
//			}
//		}
//
//		point->clusterNumber = clusterNumber;
//		point->distanceFromCluster = minDistance;
//	}
//}
//
//
//
//int cuda_add_nearest_point_to_cluster(cluster_t *clusters, int clusterCount, point_t *points, int pointCount)
//{
//	//TODO fix docu below
//	/*
//	 need to go over each point and assign to the right cluster
//    each thread can represent a point, in order for this to work
//	beside the clusters and point we will need an array of int in the size of pointCount.
//	the array will represent to which cluster the point in the id of it "i" belongs meaning
//	int belongCluster[pointCount];
//	belongCluster[0] = 4 => means point 0 belong to cluster 4
//	another array of double in the size of pointCount which will represent the "i" point distance from
//	its corrosponding cluster
//
//	 note: altohugh the threads dont need the whole cluster and only its center the entire cluster will be copy
//	as it easier and faster than extrapolating thier centers and then copy them (neglegible clusterCount should be
//	fairly small).
//
//	 number of points can be to very large therefore the division will be with 256 threads per block 
//	256 was chosen as it is divisible by the wrap size (32) 
//	
//	
//	*/
//
//	setDevice();
//
//	cluster_t *devClusters;
//	point_t *devPoints;
//	
//	// allocate GPU space
//	RETURN_IF_NOT_TRUE(allocate_GPU_space((void**)&devClusters, clusterCount, sizeof(cluster_t)));
//	RETURN_IF_NOT_TRUE(allocate_GPU_space((void**)&devPoints, pointCount, sizeof(point_t)));
//
//	//copy clusters and point to GPU's memory
//	RETURN_IF_NOT_TRUE(copy_to_GPU_memory(devClusters, clusters, clusterCount, sizeof(cluster_t)));
//	RETURN_IF_NOT_TRUE(copy_to_GPU_memory(devPoints, points, pointCount, sizeof(point_t)));
//	
//	int numBlocks = pointCount / NUM_OF_THREADS_PER_BLOCK + 1;
//	calcPointCluster <<<numBlocks, NUM_OF_THREADS_PER_BLOCK>>> (devClusters, clusterCount, devPoints, pointCount);
//	
//	RETURN_IF_NOT_TRUE(copy_from_GPU_memory(points, devPoints, pointCount, sizeof(point_t)));
//	
//	RETURN_IF_NOT_TRUE(free_GPU_space(devClusters));
//	RETURN_IF_NOT_TRUE(free_GPU_space(devPoints));
//
//	return 1;
//}
//
////int cuda_calc_new_center(cluster_t *clusters, int clusterCount)
////{
////
////	return 1;
////}
//
//
//
//__global__ void calcSumOfHelpersFromPoints(clusterHelper_t *devHelpers, point_t *devPoints, int numPoints)
//{
//	int offset = blockIdx.x * blockDim.x + threadIdx.x;
//	if (offset < numPoints)
//	{
//		point_t *point = &(devPoints[offset]);
//		clusterHelper_t *helper = &(devHelpers[point->clusterNumber]);
//		
//		double *helperVectorArg;
//		double pointVectorArg;
//		int j;
//		for (j = 0; j < NUM_OF_ELEMENTS_IN_VECTOR; j++)
//		{
//			helperVectorArg = &(helper->sumPointLocation.axis[j]);
//			pointVectorArg = point->location.axis[j];
//			atomicAdd((float*)helperVectorArg, (float)pointVectorArg);
//		}
//	}
//}
//
//int cuda_calc_partialSums(clusterHelper_t *clusterHelpers, int numOfHelpers, point_t *points, int pointCount)
//{
//
//	setDevice();
//	int numOfBlocks = pointCount / NUM_OF_THREADS_PER_BLOCK;
//
//	point_t *devPoints;
//	clusterHelper_t *devClusterHelpers;
//
//	//allocate gpu space
//	RETURN_IF_NOT_TRUE(allocate_GPU_space((void**)&devPoints, pointCount, sizeof(point_t)));
//	RETURN_IF_NOT_TRUE(allocate_GPU_space((void**)&devClusterHelpers, numOfHelpers, sizeof(clusterHelper_t)));
//	
//	//copy points to gpu
//	RETURN_IF_NOT_TRUE(copy_to_GPU_memory(devPoints, points, pointCount, sizeof(point_t)));
//
//	calcSumOfHelpersFromPoints <<<numOfBlocks, NUM_OF_THREADS_PER_BLOCK>>>(devClusterHelpers, devPoints, pointCount);
//
//	RETURN_IF_NOT_TRUE(copy_from_GPU_memory(clusterHelpers, devClusterHelpers, numOfHelpers, sizeof(clusterHelper_t)));
//
//	RETURN_IF_NOT_TRUE(free_GPU_space(devClusterHelpers));
//	RETURN_IF_NOT_TRUE(free_GPU_space(devPoints));
//
//
//	return 1;
//}