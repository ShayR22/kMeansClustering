#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h" // my header
#include <hip/hip_runtime.h> // official api header
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#define CPU_TO_GPU hipMemcpyHostToDevice
#define GPU_TO_CPU hipMemcpyDeviceToHost
#define WRAP_SIZE 32
#define NUM_OF_THREADS_PER_BLOCK 256
#define NUM_OF_POINTS_TO_GO_OVER_IN_CALC_DIAMETER_PER_THREAD 256
#define FIND_MAX_RUN_PER_POINT 10

#define RETURN_IF_NOT_TRUE(truth)if(!truth) {return 0;}

static void copy_ids_from_helpers_to_array(clusterHelper_t *helpers, int helpersCount, int *IDs);
static int find_2_power_of(int numPoints);
static void find_max_distance_for_each_point_in_helper(clusterHelper_t *helper, point_t *devPoints, double *devDiameters, int *devHelpersIDs, int *offset);
static void zero_to_the_power_of_two_complement(clusterHelper_t *helper, double *devDiameters, int *goOver);
static void zero_devArray(double *devDiameters, int arraySize);

static double find_max_from_devDiameters(double *devDiameters, int goOver);

static int setDevice();
static int allocate_GPU_space(void** data, int dataCount, int dataTypeSize);
static int free_GPU_space(void* data);
static int copy_to_GPU_memory(void* copyTo, void* copyFrom, int count, int typeSize);
static int copy_from_GPU_memory(void* copyTo, void* copyFrom, int count, int typeSize);
static int checkError(hipError_t error);

__device__ double getDistance(vector_t *v1, vector_t *v2)
{
	int i;
	double sumOfDeltaSquares = 0;
	double delta;
	for (i = 0; i < NUM_OF_ELEMENTS_IN_VECTOR; i++)
	{
		delta = v1->axis[i] - v2->axis[i];
		sumOfDeltaSquares += delta * delta;
	}
	return sqrt(sumOfDeltaSquares);
}

__device__ void advance_point_in_deltaT(point_t *p, double deltaT)
{
	int i;
	for (i = 0; i < NUM_OF_ELEMENTS_IN_VECTOR; i++)
	{
		p->location.axis[i] += p->speed.axis[i] * deltaT;
	}
}

__global__ void increment_points(point_t *devPoints, int pointsCount, double deltaT)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < pointsCount)
	{
		point_t *point = &(devPoints[index]);
		advance_point_in_deltaT(point, deltaT);
	}
}

int cuda_increment_points(point_t *points, int pointsCount, double deltaT)
{
	point_t *devPoints;
	setDevice();
	// allocate space and then copy points in to it
	RETURN_IF_NOT_TRUE(allocate_GPU_space((void**)&devPoints, pointsCount, sizeof(point_t)));
	RETURN_IF_NOT_TRUE(copy_to_GPU_memory(devPoints, points, pointsCount, sizeof(point_t)));

	int numOfBlocks = pointsCount / NUM_OF_THREADS_PER_BLOCK + 1;
	increment_points <<<numOfBlocks, NUM_OF_THREADS_PER_BLOCK >>> (devPoints, pointsCount, deltaT);
	RETURN_IF_NOT_TRUE(copy_from_GPU_memory(points, devPoints, pointsCount, sizeof(point_t)));

	RETURN_IF_NOT_TRUE(free_GPU_space(devPoints));
	return 1;
}

__global__ void calcPointNearestCluster(point_t *devPoints, int pointsCount, cluster_t *devClusters, int clustersCount)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < pointsCount)
	{
		vector_t *pointLocation = &(devPoints[index].location);
		vector_t *currentClusterLocation;

		double distance = INT_MAX;
		double currentDistance;
		int belongToo;
		int i;
		for (i = 0; i < clustersCount; i++)
		{
			currentClusterLocation = &(devClusters[i].center);
			currentDistance = getDistance(pointLocation, currentClusterLocation);
			if(currentDistance < distance)
			{
				distance = currentDistance;
				belongToo = i;
			}
		}
		devPoints[index].clusterBelongTo = belongToo;
	}
}

int cuda_add_to_each_point_its_nearest_cluster(point_t *points, int pointsCount, cluster_t *clusters, int clustersCount)
{
	setDevice();	
	cluster_t *devClusters;
	point_t *devPoints;

	// allocate GPU space
	RETURN_IF_NOT_TRUE(allocate_GPU_space((void**)&devClusters, clustersCount, sizeof(cluster_t)));
	RETURN_IF_NOT_TRUE(allocate_GPU_space((void**)&devPoints, pointsCount, sizeof(point_t)));

	//copy clusters and point to GPU's memory
	RETURN_IF_NOT_TRUE(copy_to_GPU_memory(devClusters, clusters, clustersCount, sizeof(cluster_t)));
	RETURN_IF_NOT_TRUE(copy_to_GPU_memory(devPoints, points, pointsCount, sizeof(point_t)));

	int numBlocks = pointsCount / NUM_OF_THREADS_PER_BLOCK + 1;
	calcPointNearestCluster <<<numBlocks, NUM_OF_THREADS_PER_BLOCK>>> (devPoints, pointsCount, devClusters, clustersCount);
		
	RETURN_IF_NOT_TRUE(copy_from_GPU_memory(points, devPoints, pointsCount, sizeof(point_t)));

	RETURN_IF_NOT_TRUE(free_GPU_space(devClusters));
	RETURN_IF_NOT_TRUE(free_GPU_space(devPoints));
	
	return 1;
}

__global__ void calcDiaPerOneHelperIteration(point_t *devPoints, double *devDiameters, int *devHelpersIDs, int devHelperIDsOffset, int devChunkSize, int start, int jumpSize)
{
	int offset = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (offset + start < devChunkSize)
	{
		int *IDS = &(devHelpersIDs[devHelperIDsOffset]);
		vector_t *v1 = &(devPoints[IDS[offset]].location); // extract point in the offset location
		vector_t *v2;
		double maxDiameterPointOffset = 0;
		double currentDist;

		int startLoopFrom = offset + start + 1;
		int endLoopIn = startLoopFrom + jumpSize;
		if (endLoopIn > devChunkSize)
		{
			endLoopIn = devChunkSize;
		}

		//iterate to the right of the chunkSize and get maxValue for given point
		int i;
		for (i = startLoopFrom; i < endLoopIn; i++)
		{
			v2 = &(devPoints[IDS[i]].location);

			currentDist = getDistance(v1, v2);
			if (currentDist > maxDiameterPointOffset)
			{
				maxDiameterPointOffset = currentDist;
			}
		}
		if (maxDiameterPointOffset > devDiameters[offset])
		{
			devDiameters[offset] = maxDiameterPointOffset;
		}
	}
}

__global__ void find_max_iteration_2_power_n(double *devDiameters, int halfCompares)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < halfCompares)
	{
		if (devDiameters[index] < devDiameters[2 * halfCompares - index - 1])
		{
			devDiameters[index] = devDiameters[2 * halfCompares - index - 1];
		}
	}
}

__global__ void zeroLocation(double *devDiameters, int devDiametersCount, int goOver)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x + devDiametersCount;
	if (index < goOver + devDiametersCount)
	{
		devDiameters[index] = 0;
	}
}

int cuda_calc_each_helper_diamater(point_t *points, int pointsCount, clusterHelper_t *helpers, int helpersCount)
{
		setDevice();
		point_t *devPoints;
		int *allHelpersIDs;
		int *devHelpersIDs;
		double *devDiameters;
		int devDiametersCompPower2 = find_2_power_of(pointsCount);
			
		//allocate diameters in GPU
		RETURN_IF_NOT_TRUE(allocate_GPU_space((void**)&devDiameters, devDiametersCompPower2, sizeof(double)));
		//allocate points in GPU
		RETURN_IF_NOT_TRUE(allocate_GPU_space((void**)&devPoints, pointsCount, sizeof(point_t)));
		//copy points to GPU
		RETURN_IF_NOT_TRUE(copy_to_GPU_memory(devPoints, points, pointsCount, sizeof(point_t)));
		//allocate CPU array for all helpers IDs
		RETURN_IF_NOT_TRUE((allHelpersIDs = (int*)malloc(sizeof(int) * pointsCount)) != NULL);
		//copy each helper's ids into one array
		copy_ids_from_helpers_to_array(helpers, helpersCount, allHelpersIDs);
		//allocate helperIDs in GPU
		RETURN_IF_NOT_TRUE(allocate_GPU_space((void**)&devHelpersIDs, pointsCount, sizeof(int)));
		//copy helpersIDs to GPU
		RETURN_IF_NOT_TRUE(copy_to_GPU_memory(devHelpersIDs, allHelpersIDs, pointsCount, sizeof(int)));
		
		int goOver;
		int offset = 0;
		int i;
		for (i = 0; i < helpersCount; i++)
		{
			zero_devArray(devDiameters,helpers[i].numOfPoints);
			find_max_distance_for_each_point_in_helper(&(helpers[i]), devPoints, devDiameters, devHelpersIDs, &offset);

			zero_to_the_power_of_two_complement(&(helpers[i]), devDiameters, &goOver);

			helpers[i].diameter = find_max_from_devDiameters(devDiameters, goOver);
		}

		//free CPU
		free(allHelpersIDs);
		//free CUDA 
		RETURN_IF_NOT_TRUE(free_GPU_space(devDiameters)); // free diameters
		RETURN_IF_NOT_TRUE(free_GPU_space(devPoints)); // free points
		RETURN_IF_NOT_TRUE(free_GPU_space(devHelpersIDs)); // free IDs array	
		return 1;
}

static void copy_ids_from_helpers_to_array(clusterHelper_t *helpers, int helpersCount, int *IDs)
{
	int offSetIDs = 0;
	int i;
	for (i = 0; i < helpersCount; i++)
	{
		memcpy(&(IDs[offSetIDs]), helpers[i].pointsIDs, helpers[i].numOfPoints * sizeof(int));
		offSetIDs += helpers[i].numOfPoints;
	}
}

static void find_max_distance_for_each_point_in_helper(clusterHelper_t *helper, point_t *devPoints, double *devDiameters, int *devHelpersIDs ,int *offset)
{
	int numBlocksCalcDiameters;
	int maxIteration = (helper->numOfPoints + FIND_MAX_RUN_PER_POINT - 1) / FIND_MAX_RUN_PER_POINT;
	int start = 0;
	int i;
	for (i = 0; i < maxIteration; i++)
	{
		numBlocksCalcDiameters = ((helper->numOfPoints + NUM_OF_THREADS_PER_BLOCK - 1 - start) / NUM_OF_THREADS_PER_BLOCK);
		calcDiaPerOneHelperIteration<<<numBlocksCalcDiameters, NUM_OF_THREADS_PER_BLOCK >>>(devPoints, devDiameters, devHelpersIDs, *offset, helper->numOfPoints, start, FIND_MAX_RUN_PER_POINT);
		start += FIND_MAX_RUN_PER_POINT;
	}
	*offset += helper->numOfPoints;

}

static void zero_to_the_power_of_two_complement(clusterHelper_t *helper, double *devDiameters, int *goOver)
{
	*goOver = find_2_power_of(helper->numOfPoints);
	int numPointsToZero = *goOver - helper->numOfPoints;
	if (numPointsToZero != 0)
	{
		int numBlocksForZero = (numPointsToZero + NUM_OF_THREADS_PER_BLOCK - 1) / NUM_OF_THREADS_PER_BLOCK;
		zeroLocation << <numBlocksForZero, NUM_OF_THREADS_PER_BLOCK >> > (devDiameters, helper->numOfPoints, *goOver); //O(n /(k * cudaThreads)) ~= O(1)
	}
}

static void zero_devArray(double *devDiameters,int arraySize)
{
	int numBlocksForZero = (arraySize + NUM_OF_THREADS_PER_BLOCK - 1) / NUM_OF_THREADS_PER_BLOCK;
	zeroLocation << <numBlocksForZero, NUM_OF_THREADS_PER_BLOCK >> > (devDiameters, 0, arraySize); //O(n /(k * cudaThreads)) ~= O(1)
}

static double find_max_from_devDiameters(double *devDiameters, int goOver)
{
	double result;
	int i;
	for (i = goOver / 2; i >= 1; i= i / 2)
	{
		int numBlocksForFindMax = (i + NUM_OF_THREADS_PER_BLOCK - 1) / NUM_OF_THREADS_PER_BLOCK;
		if (numBlocksForFindMax != 0)
		{
			find_max_iteration_2_power_n << <numBlocksForFindMax, NUM_OF_THREADS_PER_BLOCK >> > (devDiameters, i);
		}
		else
		{
			find_max_iteration_2_power_n << <1, NUM_OF_THREADS_PER_BLOCK >> > (devDiameters, i);
		}
	}

	copy_from_GPU_memory(&result, devDiameters, 1, sizeof(double));
	return result;
}

static int find_2_power_of(int numPoints)
{
	int x = 1;
	while (x < numPoints)
	{
		x *= 2;
	}
	return x;
}

static int setDevice()
{
	int device = 0;
	hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
	hipError_t error = hipSetDevice(device);
	return checkError(error);
}

static int allocate_GPU_space(void** data, int dataCount, int dataTypeSize)
{
	hipError_t error = hipMalloc(data, dataCount * dataTypeSize);
	return checkError(error);
}

static int free_GPU_space(void* data)
{
	hipError_t error = hipFree(data);
	return checkError(error);
}

static int copy_to_GPU_memory(void* copyTo, void* copyFrom, int count, int typeSize)
{
	hipError_t error = hipMemcpy(copyTo, copyFrom, count*typeSize, CPU_TO_GPU);
	return checkError(error);
}

static int copy_from_GPU_memory(void *copyTo, void* copyFrom, int count, int typeSize)
{
	hipError_t error = hipMemcpy(copyTo, copyFrom, count*typeSize, GPU_TO_CPU);
	return checkError(error);
}

static int checkError(hipError_t error)
{
	if (error != hipSuccess)
	{
		printf("error number %d occured\n", error);
		printf(hipGetErrorName(error));
		printf("\n");
		fflush(stdout);
		return 0;
	}
	return 1;
}
